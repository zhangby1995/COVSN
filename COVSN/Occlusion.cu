#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <time.h>
#include"PlyLoader.h"
#include <Windows.h>
#define DATA_SIZE 1024
#define NUM 512
#define checkCudaErrors(err)  __checkCudaErrors (err, __FILE__, __LINE__)
#define getLastCudaError(msg)  __getLastCudaError (msg, __FILE__, __LINE__)

int data[DATA_SIZE];

////////////////////////////////////////////////////////////////////////////////
// These are CUDA Helper functions

// This will output the proper CUDA error strings in the event that a CUDA host call returns an error


//inline void __checkCudaErrors(hipError_t err, const char *file, const int line )
//{
//    if(hipSuccess != err)
//    {
//        fprintf(stderr, "%s(%i) : CUDA Runtime API error %d: %s.\n",file, line, (int)err, hipGetErrorString( err ) );
//        OutputDebugString("");
//	 return ;        
//    }
//}
//
//// This will output the proper error string when calling hipGetLastError
//
//
//inline void __getLastCudaError(const char *errorMessage, const char *file, const int line )
//{
//    hipError_t err = hipGetLastError();
//    if (hipSuccess != err)
//    {
//        fprintf(stderr, "%s(%i) : getLastCudaError() CUDA error : %s : (%d) %s.\n",
//        file, line, errorMessage, (int)err, hipGetErrorString( err ) );
//        return ;
//    }
//}

// end of CUDA Helper Functions

//__global__ static void sumOfSquares(float * gpu_yourmother,float * gpuf,int *gpuresult,float * gpuG2)
/*__global__ static void sumOfSquares(float * gpuf,int *gpuresult,float * gpuG2)*/
__global__ static void sumOfSquares(float * gpuf, int *gpuresult)
{
	int idx = threadIdx.x;
	int tid = blockIdx.x;
	float x1 = gpuf[idx * 9];
	float y1 = gpuf[idx * 9 + 1];
	float z1 = gpuf[idx * 9 + 2];
	float x2 = gpuf[idx * 9 + 3];
	float y2 = gpuf[idx * 9 + 4];
	float z2 = gpuf[idx * 9 + 5];
	float x3 = gpuf[idx * 9 + 6];
	float y3 = gpuf[idx * 9 + 7];
	float z3 = gpuf[idx * 9 + 8];//�ڵ�����������������

	//vertex v0,v1,v2;
	//v0.x=gpuf[idx*9];
	//v0.y=gpuf[idx*9+1];
	//v0.z=gpuf[idx*9+2];
	//v1.x=gpuf[idx*9+3];
	//v1.y=gpuf[idx*9+4];
	//v1.z=gpuf[idx*9+5];
	//v2.x=gpuf[idx*9+6];
	//v2.y=gpuf[idx*9+7];
	//v2.z=gpuf[idx*9+8];//�ڵ�����������������

	//vertex E1,E2;
	//E1.x=v1.x-v0.x;
	//E1.y=v1.y-v0.y;
	//E1.z=v1.z-v0.z;
	//E2.x=v2.x-v0.x;
	//E2.y=v2.y-v0.y;
	//E2.z=v2.z-v0.z;
	/*vertex v;*/
	float x12 = x2 - x1;
	float y12 = y2 - y1;
	float z12 = z2 - z1;
	float x13 = x3 - x1;
	float y13 = y3 - y1;
	float z13 = z3 - z1;

	/*gpuG2[idx*6]=x12;
	gpuG2[idx*6+1]=y12;
	gpuG2[idx*6+2]=z12;
	gpuG2[idx*6+3]=x13;
	gpuG2[idx*6+4]=y13;
	gpuG2[idx*6+5]=z13;*/

	float x01 = gpuf[tid * 9];
	float y01 = gpuf[tid * 9 + 1];
	float z01 = gpuf[tid * 9 + 2];
	float x02 = gpuf[tid * 9 + 3];
	float y02 = gpuf[tid * 9 + 4];
	float z02 = gpuf[tid * 9 + 5];
	float x03 = gpuf[tid * 9 + 6];
	float y03 = gpuf[tid * 9 + 7];
	float z03 = gpuf[tid * 9 + 8];//���ڵ�����������������

	//vertex P1,P2,P3;
	//P1.x=y01*E2.z-z01*E2.y;
	//P1.y=z01*E2.x-x01*E2.z;
	//P1.z=x01*E2.y-y01*E2.x;

	//P2.x=y02*E2.z-z02*E2.y;
	//P2.y=z02*E2.x-x02*E2.z;
	//P2.z=x02*E2.y-y02*E2.x;

	//P3.x=y03*E2.z-z03*E2.y;
	//P3.y=z03*E2.x-x03*E2.z;
	//P3.z=x03*E2.y-y03*E2.x;
	float P1x = y01*z13 - z01*y13;
	float P1y = z01*x13 - x01*z13;
	float P1z = x01*y13 - y01*x13;

	float P2x = y02*z13 - z02*y13;
	float P2y = z02*x13 - x02*z13;
	float P2z = x02*y13 - y02*x13;

	float P3x = y03*z13 - z03*y13;
	float P3y = z03*x13 - x03*z13;
	float P3z = x03*y13 - y03*x13;


	float Qx = z1*y12 - y1*z12;
	float Qy = x1*z12 - z1*x12;
	float Qz = y1*x12 - x1*y12;

	//float det1,det2,det3;
	//det1=E1.x*P1.x+E1.y*P1.y+E1.z*E1.z;
	float t1 = (Qx*x13 + Qy*y13 + Qz*z13) / (P1x*x12 + P1y*y12 + P1z*z12);
	float u1 = (-P1x*x1 - P1y*y1 - P1z*z1) / (P1x*x12 + P1y*y12 + P1z*z12);
	float v1 = (Qx*x01 + Qy*y01 + Qz*z01) / (P1x*x12 + P1y*y12 + P1z*z12);

	float t2 = (Qx*x13 + Qy*y13 + Qz*z13) / (P2x*x12 + P2y*y12 + P2z*z12);
	float u2 = (-P2x*x1 - P2y*y1 - P2z*z1) / (P2x*x12 + P2y*y12 + P2z*z12);
	float v2 = (Qx*x02 + Qy*y02 + Qz*z02) / (P2x*x12 + P2y*y12 + P2z*z12);

	float t3 = (Qx*x13 + Qy*y13 + Qz*z13) / (P3x*x12 + P3y*y12 + P3z*z12);
	float u3 = (-P3x*x1 - P3y*y1 - P3z*z1) / (P3x*x12 + P3y*y12 + P3z*z12);
	float v3 = (Qx*x03 + Qy*y03 + Qz*z03) / (P3x*x12 + P3y*y12 + P3z*z12);
	/*int tmp=(tid*blockDim.x+idx)*6;*/
	/*
	gpu_yourmother[tmp]=t1;
	gpu_yourmother[tmp+1]=u1;
	gpu_yourmother[tmp+2]=v1;
	gpu_yourmother[tmp+3]=t2;
	gpu_yourmother[tmp+4]=u2;

	gpu_yourmother[tmp+5]=v2;
	gpu_yourmother[tmp+6]=t3;
	gpu_yourmother[tmp+7]=u3;
	gpu_yourmother[tmp+8]=v3;
	gpu_yourmother[tmp+9]=233333;
	*/
	/*gpuG2[tmp]=t1;

	gpuG2[tmp+1]=u1;
	gpuG2[tmp+2]=v1;
	gpuG2[tmp+3]=t2;
	gpuG2[tmp+4]=u2;
	gpuG2[tmp+5]=v2;*/
	if ((x1 == x01&&y1 == y01&&z1 == z01) || (x1 == x02&&y1 == y02&&z1 == z02) || (x1 == x03&&y1 == y03&&z1 == z03) ||
		(x2 == x01&&y2 == y01&&z2 == z01) || (x2 == x02&&y2 == y02&&z2 == z02) || (x2 == x03&&y2 == y03&&z2 == z03) ||
		(x3 == x01&&y3 == y01&&z3 == z01) || (x3 == x02&&y3 == y02&&z3 == z02) || (x3 == x03&&y3 == y03&&z3 == z03))
	{//����й�����Ļ�����û���ڵ�
		gpuresult[tid*blockDim.x + idx] = 1;
	}
	else
	{
		if (((t1>0) && (t1<1) && (u1>0) && (v1>0) && (u1 + v1)<1) ||
			((t2>0) && (t2<1) && (u2>0) && (v2>0) && (u2 + v2)<1) ||
			((t3>0) && (t3<1) && (u3>0) && (v3>0) && (u3 + v3)<1))//���ڵ�
		{
			gpuresult[tid*blockDim.x + idx] = 0;
		}
		else
		{
			gpuresult[tid*blockDim.x + idx] = 1;
		}
	}

	//gpuresult[j*blockDim.x+i]=(a1<0||a1>1||b1<0||b1>1||c1<0||c1>1)&&(a2<0||a2>1||b2<0||b2>1||c2<0||c2>1)&&(a3<0||a3>1||b3<0||b3>1||c3<0||c3>1);//���ڵ�
}

/*extern "C" void shelter(int camera_index,float*f, int num,int *result,float *G2)*/
extern "C" void occlusion(float *f, int num, vector<int> &result)
{
	hipSetDevice(0);
	float * gpuf;
	/*	 float * gpu_yourmother;
	float * cpu_yourmother;*/
	int * gpuresult;
	int * FF;
	/* float *gpuG2;*/
	/* float *cpuG2;*/
	FF = (int*)malloc(sizeof(int)*num*num);
	/*cpuG2=(float*)malloc(6*sizeof(float)*NUM*NUM);*/
	//cpu_yourmother=(float*)malloc(sizeof(float)*32*NUM);
	//for(int i=0;i<num;i++)
	//{
	// FF[i*num]=1;
	//}

	/*checkCudaErrors(hipMalloc((void**) &gpuf, sizeof(float)*9*NUM));*/
	hipMalloc((void**)&gpuf, sizeof(float)* 9 * num);

	//checkCudaErrors(hipMalloc((void**) &gpu_yourmother, sizeof(float)*32*NUM));
	/* checkCudaErrors(hipMalloc((void**) &gpuG2, sizeof(float)*6*NUM*NUM));*/

	/*checkCudaErrors(hipMalloc((void**) &gpuresult, sizeof(int)*NUM*NUM));*/
	hipMalloc((void**)&gpuresult, sizeof(int)*num*num);

	/*checkCudaErrors(hipMemcpy(gpuf, f, sizeof(float)*9*num,hipMemcpyHostToDevice));*/
	hipMemcpy(gpuf, f, sizeof(float)* 9 * num, hipMemcpyHostToDevice);

	//sumOfSquares<<<num ,num>>>(gpu_yourmother,gpuf,gpuresult,gpuG2);
	/*sumOfSquares<<<num ,num>>>(gpuf,gpuresult,gpuG2);*/
	sumOfSquares << <num, num >> >(gpuf, gpuresult);

	/* checkCudaErrors(hipMemcpy(FF, gpuresult, sizeof(int)*num*num, hipMemcpyDeviceToHost));*/
	hipMemcpy(FF, gpuresult, sizeof(int)*num*num, hipMemcpyDeviceToHost);

	/*checkCudaErrors(hipMemcpy(cpuG2, gpuG2, sizeof(float)*6*num*num, hipMemcpyDeviceToHost));*/
	//checkCudaErrors(hipMemcpy(cpu_yourmother, gpu_yourmother, sizeof(float)*32*num, hipMemcpyDeviceToHost));
	hipFree(gpuf);
	/*hipFree(gpuG2);*/
	hipFree(gpuresult);
	//hipFree(gpu_yourmother);
	for (int s = 0; s<num; s++)
	{
		for (int t = 0; t<num; t++)
		{
			FF[s*num] = FF[s*num + t] && FF[s*num];
		}
		result[s] = result[s] && FF[s*num];
	}
	if (FF)
	{
		delete FF;
		FF = NULL;
	}
}


